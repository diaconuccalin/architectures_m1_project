#include "hip/hip_runtime.h"
//
// Created by calin on 22-Dec-23.
//

#include <cstdio>

extern "C" {
#include "cu_bellman_ford.cuh"
}


void error_handling() {
    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
        printf("%s\n", hipGetErrorString(error));
}


void cu_initialize_single_source(graph *G, node *s) {
    for (int i = 0; i < G->n; i++) {
        G->nodes[i].d = INT_MAX;
        G->nodes[i].pi = nullptr;
    }

    s->d = 0;
}


__global__ void cu_relax(
        int *m,
        int *nodes_ds,
        int *nodes_pis,
        int *edges_sources,
        int *edges_destinations,
        int *edges_weights) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < *m) {
        int source_index = edges_sources[index];
        int destination_index = edges_destinations[index];

        if (nodes_ds[source_index] < INT_MAX) {
            if (nodes_ds[destination_index] > (nodes_ds[source_index] + edges_weights[index])) {
                nodes_ds[destination_index] = nodes_ds[source_index] + edges_weights[index];
                nodes_pis[destination_index] = source_index;
            }
        }
    }
}


extern "C"
bool cu_bellman_ford(graph *G, node *s) {
    cu_initialize_single_source(G, s);
    double start_time = get_time();

    // Size information
    int *d_n, *d_m;

    hipMalloc((void **) &d_n, sizeof(int));
    hipMalloc((void **) &d_m, sizeof(int));
    error_handling();

    hipMemcpy(d_n, &G->n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_m, &G->m, sizeof(int), hipMemcpyHostToDevice);
    error_handling();

    // Nodes information
    int *nodes_ds, *nodes_pis;
    int *d_nodes_ds, *d_nodes_pis;

    hipMalloc((void **) &d_nodes_ds, G->n * sizeof(int));
    hipMalloc((void **) &d_nodes_pis, G->n * sizeof(int));
    error_handling();

    hipHostMalloc((void **) &nodes_ds, G->n * sizeof(int));
    hipHostMalloc((void **) &nodes_pis, G->n * sizeof(int));
    error_handling();

    for (int i = 0; i < G->n; i++) {
        nodes_ds[i] = G->nodes[i].d;
        nodes_pis[i] = -1;
    }

    hipMemcpy(d_nodes_ds, nodes_ds, G->n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes_pis, nodes_pis, G->n * sizeof(int), hipMemcpyHostToDevice);
    error_handling();

    // Edges information
    int *edges_sources, *edges_destinations, *edges_weights;
    int *d_edges_sources, *d_edges_destinations, *d_edges_weights;

    hipMalloc((void **) &d_edges_sources, G->m * sizeof(int));
    hipMalloc((void **) &d_edges_destinations, G->m * sizeof(int));
    hipMalloc((void **) &d_edges_weights, G->m * sizeof(int));
    error_handling();

    hipHostMalloc((void **) &edges_sources, G->m * sizeof(int));
    hipHostMalloc((void **) &edges_destinations, G->m * sizeof(int));
    hipHostMalloc((void **) &edges_weights, G->m * sizeof(int));
    error_handling();

    for (int i = 0; i < G->m; i++) {
        edges_sources[i] = find_node_id_by_name(G->edges[i].source->name, G->nodes, G->m);
        edges_destinations[i] = find_node_id_by_name(G->edges[i].destination->name, G->nodes, G->m);
        edges_weights[i] = G->edges[i].weight;
    }

    hipMemcpy(d_edges_sources, edges_sources, G->m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges_destinations, edges_destinations, G->m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges_weights, edges_weights, G->m * sizeof(int), hipMemcpyHostToDevice);
    error_handling();

    hipDeviceSynchronize();

    double end_time = get_time();
    printf("\nMemory allocation time: %fs.\n", end_time - start_time);

    // Processing
    dim3 grid_dim(int(sqrt(G->m)));
    dim3 block_dim(G->m / grid_dim.x + 1);

    start_time = get_time();
    for (int i = 1; i < G->n; i++) {
        cu_relax<<<grid_dim, block_dim>>>(
                d_m,
                d_nodes_ds,
                d_nodes_pis,
                d_edges_sources,
                d_edges_destinations,
                d_edges_weights
        );
        error_handling();
        hipDeviceSynchronize();
    }
    end_time = get_time();
    printf("CUDA execution time: %fs.\n", end_time - start_time);

    hipMemcpy(nodes_ds, d_nodes_ds, G->n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(nodes_pis, d_nodes_pis, G->n * sizeof(int), hipMemcpyDeviceToHost);
    error_handling();
    hipDeviceSynchronize();

    for (int i = 0; i < G->n; i++) {
        G->nodes[i].d = nodes_ds[i];
        G->nodes[i].pi = G->nodes[nodes_pis[i]].name;
    }

    hipFree(d_n);
    hipFree(d_m);
    hipFree(d_nodes_ds);
    hipFree(d_nodes_pis);
    hipFree(d_edges_sources);
    hipFree(d_edges_destinations);
    hipFree(d_edges_weights);
    error_handling();
    hipDeviceSynchronize();

    hipHostFree(nodes_ds);
    hipHostFree(nodes_pis);
    hipHostFree(edges_sources);
    hipHostFree(edges_destinations);
    hipHostFree(edges_weights);
    error_handling();

    for (int i = 0; i < G->m; i++) {
        edge *current_edge = &G->edges[i];
        node *u = current_edge->source;
        node *v = current_edge->destination;

        if (v->d > (u->d + current_edge->weight))
            return false;
    }

    return true;
}
